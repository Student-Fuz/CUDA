#include <hipDNN.h>
#include "alchemy.h"

using namespace std;
using namespace alchemy;

int main()
{
    // image
    auto image = imread("red.png");
    auto image_float = Matrix32f(image);

    //handle
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    // input
    Tensor<float> input({ 1, image.channels(), image.rows, image.cols });
    Memory::copy(image_float.count() * sizeof(float), input.gptr(), image_float.ptr());

    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor,
                               HIPDNN_TENSOR_NHWC,
                               HIPDNN_DATA_FLOAT,
                               input.shape(0), input.shape(1), input.shape(2), input.shape(3));

    // output
    Tensor<float> output(input.shape());
    vector_set_gpu(output.count(), 0.0f, output.gptr());

    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor,
                               HIPDNN_TENSOR_NHWC,
                               HIPDNN_DATA_FLOAT,
                               output.shape(0), output.shape(1), output.shape(2), output.shape(3));

    // kernel
    Tensor<float> kernel({ output.shape(1), input.shape(1), 3, 3 });
    auto kernel_size = kernel.count(2, 4);
    float kernel_[kernel_size] = { 0, 1, 0, 1, -4, 1, 0, 1, 0 };
    for(auto i = 0; i < kernel.count(0, 2); ++i) {
        memcpy(kernel.cptr() + i * kernel_size, kernel_, kernel_size * sizeof(float));
    }

    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor,
                               HIPDNN_DATA_FLOAT,
                               HIPDNN_TENSOR_NCHW,
                               kernel.shape(0), kernel.shape(1), kernel.shape(2), kernel.shape(3));
    // convolution descriptor
    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnSetConvolution2dDescriptor(conv_descriptor,
                                    1, 1, // zero-padding
                                    1, 1, // stride
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(handle,
                                        input_descriptor,
                                        kernel_descriptor,
                                        conv_descriptor,
                                        output_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        0,
                                        &algo);

    // workspace size && allocate memory
    size_t workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                            input_descriptor,
                                            kernel_descriptor,
                                            conv_descriptor,
                                            output_descriptor,
                                            algo,
                                            &workspace_size);

    void * workspace = nullptr;
    hipMalloc(&workspace, workspace_size);

    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(handle,
                            &alpha, input_descriptor, input.gptr(),
                            kernel_descriptor, kernel.gptr(),
                            conv_descriptor, algo,
                            workspace, workspace_size,
                            &beta, output_descriptor, output.gptr());

    Matrix32f output_image(image.shape());
    hipMemcpy(output_image.ptr(), output.gptr(), image.count() * sizeof(float), hipMemcpyDeviceToHost);

    // destroy
    hipFree(workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);

    hipdnnDestroy(handle);


    // show
    imshow("original", image);
    imshow("output", Matrix(output_image/3.0));

    waitKey(0);
    return 0;
}